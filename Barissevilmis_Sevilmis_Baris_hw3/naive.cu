
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <stdint.h>
#include <immintrin.h>
#include <cassert>
using namespace std;


void usage()
{
  cout << "USAGE: ./exec <filename> <device>" << endl;
  exit(0);
}

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}
__global__
void PreProcess(double* xCuda, double* xx, int N)
{
  long long int ind = ((blockDim.x * blockIdx.x + threadIdx.x) * N);
  double* my_xCuda = xCuda + ind;
  
  for(int j = 0; j < N; j++)
    my_xCuda[j] = xx[j];
}
__global__
void PermanentCalculator(long long int K, double* xCuda, double* pCuda, int* my_MCuda, long long int CHUNK, long long int N)
{
  long long int index = ((blockDim.x * blockIdx.x + threadIdx.x) * CHUNK);

  if(index < K)
    {
      long long int ind = ((blockDim.x * blockIdx.x + threadIdx.x) * N);
      long long int START = index + 1;
      long long int END = index + CHUNK + 1;
      long long int yy = index ^ (index >> 1LL);
      long long int y, y_prev;
      long long int FFS, z;
      double s, prodSign;
      double* my_xCuda = xCuda + ind;
      double pSelf = 0.0;
      
      long long int temp_y = yy;      
      long long int BPC = __popcll(yy);
      for(int n = 0; n < BPC; n++)
	{
	  FFS = __ffsll(temp_y) - 1;
	  temp_y &= ~(1LL << FFS);
	  for(int m = 0; m < N; m++)
	    {
	      my_xCuda[m] += my_MCuda[m + (FFS*N)];
	    }
	}
      prodSign = ((index + 1) & 1LL) ? -1.0 : 1.0;
      for(long long int ii = START; (ii < END) && (ii < K); ii++)
	{
	  y = (ii ^ (ii >> 1LL));
	  y_prev = (ii - 1) ^ ((ii - 1) >> 1LL);
	  z = __ffsll(y ^ y_prev) - 1;      
	  s = ((y >> z) & 1LL) ? 1.0 : -1.0;
	  double temp = 1.0;
	  for(int jj = 0; jj < N; jj++)
	    {
	      my_xCuda[jj] += (s * my_MCuda[jj + (z * N)]); 
	      temp *= my_xCuda[jj];
	    }
	  pSelf += (prodSign * temp);
	  prodSign *= -1.0;
	  
	}
      atomicAdd(pCuda, pSelf);
    }
}


int main(int argc, const char** argv)
{

  if(argc != 3)
    usage();

  string line;

  const char* filename = argv[1];
  ifstream input (filename);
  if(input.fail())
    return 0;

  int cudaDevice = atoi(argv[2]);
  hipSetDevice(cudaDevice);

  hipDeviceProp_t prop;
  hipGetDeviceProperties(&prop, cudaDevice);
  
  long long int N;
  long long int Nsqr;
  int **M;
  int *Mrow;
  int *MCuda;
  getline(input,line);
  N = atoi(line.c_str());
  Nsqr = N*N;
  checkCuda(hipMalloc((void**)&MCuda, sizeof(int)*Nsqr));

  
  Mrow = (int*)(malloc(sizeof(int)*Nsqr));
  M = (int**)(malloc(sizeof(int*)*N));
  for(int i = 0; i < N; i ++)
    {
      M[i] = (int*)(malloc(sizeof(int)*N));
    }

  int linectr = 0;
  while(getline(input,line))
    {
      stringstream ss(line);
      int temp;
      int ctr = 0;
      while(ss >> temp)
	{
	  M[linectr][ctr++] = temp;
	}
      linectr++;
    }

  int trctr = 0;
  for(int i = 0; i < N; i++)
    for(int j = 0; j < N; j++)
      Mrow[trctr++] = M[j][i];
	 
  hipMemcpy(MCuda, Mrow, sizeof(int)*Nsqr, hipMemcpyHostToDevice);
  
  double start, end, initt;
  ////YOUR CODE GOES HERE
  long long int K;
  K = 1LL << (N-1);
  double result = 0;
  
  double *p = (double*)(malloc(sizeof(double)));
  *p = 1.0;

  double *x = (double*)(malloc(sizeof(double)*N));
  double *sumCol = (double*)(malloc(sizeof(double)*N)); 
  double *lastCol = (double*)(malloc(sizeof(double)*N));
  memset(sumCol, 0, sizeof(double)*N);
  memset(lastCol, 0, sizeof(double)*N);

  //Decide on size given K
  //dim3 GRID(1024);
  //dim3 BLOCK(32);
  //long long int TOTAL = BLOCK.x*GRID.x;
  long long int GRID = 1024;
  long long int BLOCK = 128;
  long long int TOTAL = BLOCK * GRID;
  
  long long int CHUNK = (K + (TOTAL-1)) / TOTAL;
  long long int xSIZE = TOTAL * N;

  double *pCuda;  
  double *xCuda;
  double *xx;
  hipMalloc((void**)&pCuda, sizeof(double));
  hipMalloc((void**)&xx, sizeof(double)*N);
  hipMalloc((void**)&xCuda, sizeof(double)*xSIZE);
  //printf("Device Variable Copying:\t%s\n", cudaGetErrorString(cudaGetLastError()));

  //BEGINNING
  initt = omp_get_wtime();

  for(int i = 0; i< N; i++)
    {
      lastCol[i] = M[i][N-1];
      for(int j = 0; j < N; j++)
	sumCol[i] += M[i][j];
      x[i] = (lastCol[i] - sumCol[i]/2);
      (*p) *= x[i];
    }
      
  hipMemset(xCuda, 0, sizeof(double)* xSIZE);
  hipMemcpy(xx, x, sizeof(double)* N, hipMemcpyHostToDevice);
  hipMemcpy(pCuda, p, sizeof(double), hipMemcpyHostToDevice);
  
  start = omp_get_wtime();
  PreProcess<<<GRID,BLOCK>>>(xCuda, xx, N);
  hipDeviceSynchronize();
  PermanentCalculator<<<GRID,BLOCK>>>(K, xCuda, pCuda, MCuda, CHUNK, N);
  hipDeviceSynchronize();
  hipMemcpy(p, pCuda, sizeof(double), hipMemcpyDeviceToHost);
  
  result = (4 * (N & 1) - 2) * (*p);
  
  //ENDING
  end = omp_get_wtime();
  
  //cout << "Threads:" << TOTAL << "\tResult:" << result << "\tTime:" << end - start << "s" << "\tTotal Time:"<< end - initt << endl;
  cout << TOTAL << "," << result << "," << end - initt << "\n";
  
  for(int i = 0; i < N; i++)
    free(M[i]);

  free(p);
  free(x);
  free(sumCol);
  free(lastCol);
  hipFree(xx);
  hipFree(xCuda);
  hipFree(pCuda);
  hipFree(MCuda);
  free(M);
  free(Mrow);
  
  return 0;
}

      
