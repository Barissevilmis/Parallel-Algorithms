
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <stdlib.h>
#include <vector>
#include <stdio.h>
#include <string.h>
#include <queue>
#include <algorithm>
#include <cassert>

using namespace std;

void usage()
{
  cout << "USAGE: ./exec <filename> <deviceNum>" << endl;
  exit(0);
}

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void ConvertToCRS(int* & target_ind, int* & start_ind, const vector<vector<int> > &adj_list, int RS)
{
  int init = 0;
  for(int ii = 0; ii < RS ; ii++)
    { 
      start_ind[ii] = init;
      int siz = adj_list[ii].size();
      int where = 0;
      for(int jj = init; jj < init + siz; jj++)
	target_ind[jj] = adj_list[ii][where++];
      init += siz;
    }
  start_ind[RS] = init;
}
__global__
void INIT(int* target_ind, int* start_ind, int* results, unsigned int startNode, int* SIZE)
{

  int tid = threadIdx.x;
  int v = results[startNode];
  int st = start_ind[v];
  results[target_ind[st+tid]] = 1;
  atomicSub(SIZE,1);
  
}
__global__
void Check_BFS(int* results, int RS, int* SIZE, unsigned int LIMIT, unsigned int TOTAL)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  for(int jj = index; jj < RS; jj+=TOTAL)
    {
      if(results[jj] == -1)
	{
	  atomicAdd(SIZE,1);
	}
    }
}

__global__
void BFS_Top_Down(int* target_ind, int* start_ind, int* results, int v, int RS, unsigned int TOTAL, int* SIZE)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;


  for(int jj = index; jj < RS; jj+=TOTAL)
    {
      if(results[jj] == v)
	{
	  atomicAdd(SIZE,1);
	  int start_loc = start_ind[jj];
	  int end_loc = start_ind[jj + 1];
	  int curr = results[jj];
	  for(int ii = start_loc; ii < end_loc; ii++)
	    {
	      if(results[target_ind[ii]] == -1)
		{
		  results[target_ind[ii]] = curr + 1;
		}
	   
	    }
	}
    }
}


void BFS(int* & target_ind, int* & start_ind, int* & results, unsigned int RS, unsigned int NNS, unsigned int startNode)
{
  results[startNode] = 0;
  int v = 0;
  int *res;
  int *target_arr;
  int *start_arr;
  checkCuda(hipMalloc((void **)&res, sizeof(int)*RS));
  checkCuda(hipMalloc((void **)&target_arr, sizeof(int)*NNS));
  checkCuda(hipMalloc((void **)&start_arr, sizeof(int)*(RS+1)));
  //printf("Device Variable Copying:\t%s\n", cudaGetErrorString(cudaGetLastError()));

  int *SIZE;
  int *size = (int*)(malloc(sizeof(int)));
  int *zero = (int*)(malloc(sizeof(int)));
  unsigned int CHUNK_SIZE = 1024;
  unsigned int LIMIT = (RS+CHUNK_SIZE-1) / CHUNK_SIZE;
  unsigned int TOTAL = LIMIT*CHUNK_SIZE;
  dim3 GRID(LIMIT);
  dim3 BLOCK(CHUNK_SIZE);
  *size = 1;
  *zero = 0;
  //cudaMallocHost(&size,sizeof(int));
  checkCuda(hipMalloc((void**)&SIZE,sizeof(int)));

  checkCuda(hipMemcpy(res, results, sizeof(int)*RS,hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(start_arr, start_ind, sizeof(int)*(RS+1),hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(target_arr, target_ind, sizeof(int)*NNS,hipMemcpyHostToDevice));
 
  double start = omp_get_wtime();
  while(*size > 0)
    {
      hipMemcpy(SIZE, zero, sizeof(int), hipMemcpyHostToDevice);
      BFS_Top_Down<<<GRID,BLOCK>>>(target_arr, start_arr, res, v, RS, TOTAL, SIZE);
      hipDeviceSynchronize();
      hipMemcpy(size, SIZE, sizeof(int),hipMemcpyDeviceToHost);      
      v++;
      //Check_BFS<<<GRID,BLOCK>>>(res, RS, SIZE, LIMIT, TOTAL);
      //cudaDeviceSynchronize();
      //printf("%d-%d-%d-%d\n",*size,v,results[startNode],start_ind[target_ind[start_ind[startNode]]+1]-start_ind[target_ind[start_ind[startNode]]]);
    }
  
  double end = omp_get_wtime();
  cout << "\tSource:" << startNode << "\tTime:" << end - start << " s" << endl;
  checkCuda(hipMemcpy(results, res, sizeof(int)*RS,hipMemcpyDeviceToHost));

  free(size);
  free(zero);
  hipFree(SIZE);
  hipFree(res);
  hipFree(target_arr);
  hipFree(start_arr);
}

int main(int argc, const char** argv)
{

  if(argc != 3)
    usage();

  
  int devId = 0;
  devId = atoi(argv[2]); 
  hipSetDevice(devId); 
  string line;
  vector<vector<int> > adj_list;
  //See which node you are on by index+min
  
  const char* filename = argv[1];
  
  ifstream input (filename);
  if(input.fail())
    return 0;

  //Find first node
  unsigned int mini = 10000000;
  while(getline(input,line))
    {
      stringstream sf(line);
      int temp;
      if(line.find('%') != string::npos)
	continue;
      else
	{
	  while(sf >> temp)
	    {
	      if(temp < mini)
		mini = temp;
	    }
	}
    }
  //Set to begin
  input.clear();
  input.seekg(0, ios::beg);
  int ctr = 0;
  bool mode = false;
  unsigned int NNS, RS;
  while(getline(input,line))
    {
      stringstream ss(line);
      int temp, temp2, temp3;
      if(line.find("%%") != string::npos && line.find(" symmetric") != string::npos)
	mode = true;
      else if(mode)
	{
	  if(line.find('%') != string::npos)
	    continue;
	  //First line containing row number, column number and NNS
	  else if(!ctr)
	    {
	      ss >> temp >> temp2 >> temp3;
	      NNS = temp3;
	      adj_list.resize(temp);
	      RS = temp;
	      ctr++;
	    }
      //Lines containing start and end of an edge
	  else if(ctr)
	    {
	      ss >> temp >> temp2;
	      if(temp != temp2)
		{
		  adj_list[temp - mini].push_back(temp2 - mini);
		  adj_list[temp2 - mini].push_back(temp - mini);
		}
	    }
	}
      //Get past comment lines
      else if(!mode)
	{
	  if(line.find('%') != string::npos)
	    continue;
	  //First line containing row number, column number and NNS
	  else if(!ctr)
	    {
	      ss >> temp >> temp2 >> temp3;
	      NNS = temp3;
	      adj_list.resize(temp);
	      RS = temp;
	      ctr++;
	    }
	  //Lines containing start and end of an edge
	  else if(ctr)
	    {
	      ss >> temp >> temp2;
	      if(temp != temp2)
		adj_list[temp - mini].push_back(temp2 - mini);
	    }
	}
    }

  //Remove duplicates
  NNS = 0;
  for(int i = 0; i < adj_list.size(); i++)
    {
      sort(adj_list[i].begin(), adj_list[i].end());
      adj_list[i].erase(unique(adj_list[i].begin(), adj_list[i].end()), adj_list[i].end());
      NNS += adj_list[i].size();
    }
  
  int* target_ind = (int*)(malloc(sizeof(int)*NNS));
  //cudaMallocHost(&target_ind, sizeof(int)*NNS);
  int* start_ind = (int*)(malloc(sizeof(int)*(RS+1)));
  //cudaMallocHost(&start_ind, sizeof(int)*(RS+1));
  int* results = (int*)(malloc(sizeof(int)*RS));
  //cudaMallocHost(&results, sizeof(int)*RS);
 
  memset(target_ind, 0, sizeof(int)* NNS);
  memset(start_ind, 0, sizeof(int)* (RS+1));
  memset(results, -1, sizeof(int)* RS);
  
  ConvertToCRS(target_ind, start_ind, adj_list, RS);
  
  //Start Node as parameter
  cout << "Graph converted to 0-base(Initial node is 0)\n";
  unsigned int startNode;
  //cout << "Please enter the start node: ";
  //cin >> startNode;
  //cout << endl;
  startNode = 0;
  
  BFS(target_ind, start_ind, results, RS, NNS, startNode);

  ofstream myfile;
  myfile.open("cudaresults.txt");
  for(int i = 0; i < RS; i++)
    myfile<< results[i] <<"\n";
  myfile.close();
  
  hipDeviceSynchronize();
  //cudaFreeHost(target_ind);
  //cudaFreeHost(start_ind);
  //cudaFreeHost(results);
  free(target_ind);
  free(results);
  free(start_ind);
  
  return 0;
}
