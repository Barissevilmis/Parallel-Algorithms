
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <stdlib.h>
#include <vector>
#include <stdio.h>
#include <string.h>
#include <queue>
#include <algorithm>
#include <cassert>

using namespace std;

void usage()
{
  cout << "USAGE: ./exec <filename> <deviceNum>" << endl;
  exit(0);
}

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void ConvertToCRS(int* & target_ind, int* & start_ind, const vector<vector<int> > &adj_list, int RS)
{
  int init = 0;
  for(int ii = 0; ii < RS ; ii++)
    { 
      start_ind[ii] = init;
      int siz = adj_list[ii].size();
      int where = 0;
      for(int jj = init; jj < init + siz; jj++)
	target_ind[jj] = adj_list[ii][where++];
      init += siz;
    }
  start_ind[RS] = init;
}
__global__
void INIT(int* target_ind, int* start_ind, int* results, unsigned int startNode, int* SIZE)
{

  int tid = threadIdx.x;
  int v = results[startNode];
  int st = start_ind[v];
  results[target_ind[st+tid]] = 1;
  atomicSub(SIZE,1);
  
}
__global__
void Check_BFS(int* results, int RS, int* SIZE, int* switchCtr, unsigned int TOTAL, int v)
{
  int index = blockDim.x * blockIdx.x + threadIdx.x;

  for(int jj = index; jj < RS; jj+=TOTAL)
    {
      if(results[jj] == -1)
	{
	  atomicAdd(SIZE,1);
	}
      else if(results[jj] == v)
	{
	  atomicAdd(switchCtr,1);
	}
    }
}

__global__
void BFS_Top_Down(int* target_ind, int* start_ind, int* results, int v, int RS, unsigned int TOTAL, int* SIZE)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  for(int jj = index; jj < RS; jj+=TOTAL)
    {
      if(results[jj] == v)
	{
	  atomicAdd(SIZE, 1);
	  int start_loc = start_ind[jj];
	  int end_loc = start_ind[jj + 1];
	  int curr = results[jj];
	  for(int ii = start_loc; ii < end_loc; ii++)
	    {
	      if(results[target_ind[ii]] == -1)
		{
		  results[target_ind[ii]] = curr + 1;
		}
	   
	    }
	}
    }
}

__global__
void BFS_Bottom_Up(int* target_ind, int* start_ind, int* results, int v, unsigned int RS, unsigned int TOTAL, int* SIZE)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  for(int jj = index; jj < RS; jj+=TOTAL)
    {
      if(results[jj] == -1)
	{
	  int start_loc = start_ind[jj];
	  int end_loc = start_ind[jj + 1];
	  int target;
	  bool raviolli = false;
	  for(int ii = start_loc; ii < end_loc && raviolli == false; ii++)
	    {
	      target = results[target_ind[ii]];
	      if(target == v)
		{
		  results[jj] = target + 1;
		  atomicAdd(SIZE, 1);
		  raviolli = true;
		}
	    }
	}
    }
}
__global__
void BFS_Bottom_Up_Directed(int* target_rev, int* start_rev, int* results, int v, unsigned int RS, unsigned int TOTAL, int* SIZE)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  for(int jj = index; jj < RS; jj+=TOTAL)
    {
      if(results[jj] == -1)
	{
	  int start_loc = start_rev[jj];
	  int end_loc = start_rev[jj + 1];
	  int target;
	  bool raviolli = false;
	  for(int ii = start_loc; ii < end_loc && raviolli == false; ii++)
	    {
	      target = results[target_rev[ii]];
	      if(target == v)
		{
		  results[jj] = target + 1;
		  atomicAdd(SIZE, 1);
		  raviolli = true;
		}
	    }
	}
    }
}
void BFS(int* & target_ind, int* & start_ind, int* & results, unsigned int RS, unsigned int NNS, unsigned int startNode)
{
  results[startNode] = 0;
  int v = 0;
  int *res;
  int *target_arr;
  int *start_arr;
  checkCuda(hipMalloc((void **)&res, sizeof(int)*RS));
  checkCuda(hipMalloc((void **)&target_arr, sizeof(int)*NNS));
  checkCuda(hipMalloc((void **)&start_arr, sizeof(int)*(RS+1)));
  //printf("Device Variable Copying:\t%s\n", cudaGetErrorString(cudaGetLastError()));

  int *SIZE;
  int switchCtr = RS/32;
  int *size = (int*)(malloc(sizeof(int)));
  int *zero = (int*)(malloc(sizeof(int)));
  unsigned int CHUNK_SIZE = 1024;
  unsigned int LIMIT = (RS+CHUNK_SIZE-1) / CHUNK_SIZE;
  unsigned int TOTAL = LIMIT*CHUNK_SIZE;
  dim3 GRID(LIMIT);
  dim3 BLOCK(CHUNK_SIZE);
  *size = 1;
  *zero = 0;
  //cudaMallocHost(&size,sizeof(int));
  checkCuda(hipMalloc((void**)&SIZE,sizeof(int)));
  checkCuda(hipMemcpy(res, results, sizeof(int)*RS,hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(start_arr, start_ind, sizeof(int)*(RS+1),hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(target_arr, target_ind, sizeof(int)*NNS,hipMemcpyHostToDevice));
 
  double start = omp_get_wtime();
  while(*size > 0)
    {
      hipMemcpy(SIZE, zero, sizeof(int), hipMemcpyHostToDevice);
      if(*size < switchCtr)
	{
	  BFS_Top_Down<<<GRID,BLOCK>>>(target_arr, start_arr, res, v, RS, TOTAL, SIZE);
	  hipDeviceSynchronize();
	}
      else
	{
	  BFS_Bottom_Up<<<GRID,BLOCK>>>(target_arr, start_arr, res, v, RS, TOTAL, SIZE);
	  hipDeviceSynchronize();
	}      
      //*switchCtr = 0;
      //Check_BFS<<<GRID,BLOCK>>>(res, RS, SIZE, SWITCH, TOTAL, v);
      //cudaDeviceSynchronize();
      hipMemcpy(size, SIZE, sizeof(int),hipMemcpyDeviceToHost);
      v++;
    }
  
  double end = omp_get_wtime();
  cout << "\tSource:" << startNode << "\tTime:" << end - start << " s" << endl;
  checkCuda(hipMemcpy(results, res, sizeof(int)*RS,hipMemcpyDeviceToHost));

  free(size);
  free(zero);
  hipFree(SIZE);
  hipFree(res);
  hipFree(target_arr);
  hipFree(start_arr);
}

void BFS_Directed(int* & target_ind, int* & start_ind, int* & target_rev, int* & start_rev, int* & results, unsigned int RS, unsigned int NNS, unsigned int NNSrev, unsigned int startNode)
{
  results[startNode] = 0;
  int v = 0;
  int *res;
  int *target_rev_arr;
  int *start_rev_arr;
  int *target_arr;
  int *start_arr;
  checkCuda(hipMalloc((void **)&res, sizeof(int)*RS));
  checkCuda(hipMalloc((void **)&target_arr, sizeof(int)*NNS));
  checkCuda(hipMalloc((void **)&start_arr, sizeof(int)*(RS+1)));
  checkCuda(hipMalloc((void **)&target_rev_arr, sizeof(int)*NNSrev));
  checkCuda(hipMalloc((void **)&start_rev_arr, sizeof(int)*(RS+1)));
  //printf("Device Variable Copying:\t%s\n", cudaGetErrorString(cudaGetLastError()));

  int *SIZE;
  int switchCtr = RS/32;
  int *size = (int*)(malloc(sizeof(int)));
  int *zero = (int*)(malloc(sizeof(int)));
  unsigned int CHUNK_SIZE = 1024;
  unsigned int LIMIT = (RS+CHUNK_SIZE-1) / CHUNK_SIZE;
  unsigned int TOTAL = LIMIT*CHUNK_SIZE;
  dim3 GRID(LIMIT);
  dim3 BLOCK(CHUNK_SIZE);
  *size = 1;
  *zero = 0;
  //cudaMallocHost(&size,sizeof(int));
  checkCuda(hipMalloc((void**)&SIZE,sizeof(int)));
  checkCuda(hipMemcpy(res, results, sizeof(int)*RS,hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(start_arr, start_ind, sizeof(int)*(RS+1),hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(target_arr, target_ind, sizeof(int)*NNS,hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(start_rev_arr, start_rev, sizeof(int)*(RS+1),hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(target_rev_arr, target_rev, sizeof(int)*NNSrev,hipMemcpyHostToDevice));
 
  double start = omp_get_wtime();
  while(*size > 0)
    {
      hipMemcpy(SIZE, zero, sizeof(int), hipMemcpyHostToDevice);
      if(*size < switchCtr)
	{
	  BFS_Top_Down<<<GRID,BLOCK>>>(target_arr, start_arr, res, v, RS, TOTAL, SIZE);
	  hipDeviceSynchronize();
	}
      else
	{
	  BFS_Bottom_Up_Directed<<<GRID,BLOCK>>>(target_rev_arr, start_rev_arr, res, v, RS, TOTAL, SIZE);
	  hipDeviceSynchronize();
	}      
      //*switchCtr = 0;
      //Check_BFS<<<GRID,BLOCK>>>(res, RS, SIZE, SWITCH, TOTAL, v);
      //cudaDeviceSynchronize();
      v++;
      hipMemcpy(size, SIZE, sizeof(int),hipMemcpyDeviceToHost);
    }
  
  double end = omp_get_wtime();
  cout << "\tSource:" << startNode << "\tTime:" << end - start << " s" << endl;
  checkCuda(hipMemcpy(results, res, sizeof(int)*RS,hipMemcpyDeviceToHost));

  free(size);
  free(zero);
  hipFree(SIZE);
  hipFree(res);
  hipFree(target_rev_arr);
  hipFree(start_rev_arr);
  hipFree(target_arr);
  hipFree(start_arr);
}


int main(int argc, const char** argv)
{

  if(argc != 3)
    usage();

  
  int devId = 0;
  devId = atoi(argv[2]); 
  hipSetDevice(devId); 
  string line;
  vector<vector<int> > adj_list;
  vector<vector<int> > rev_list;
  //See which node you are on by index+min
  
  const char* filename = argv[1];
  
  ifstream input (filename);
  if(input.fail())
    return 0;

  //Find first node
  unsigned int mini = 10000000;
  while(getline(input,line))
    {
      stringstream sf(line);
      int temp;
      if(line.find('%') != string::npos)
	continue;
      else
	{
	  while(sf >> temp)
	    {
	      if(temp < mini)
		mini = temp;
	    }
	}
    }
  //Set to begin
  input.clear();
  input.seekg(0, ios::beg);
  int ctr = 0;
  bool mode = false;
  unsigned int NNS, RS, NNSrev;
  while(getline(input,line))
    {
      stringstream ss(line);
      int temp, temp2, temp3;
      if(line.find("%%") != string::npos && line.find(" symmetric") != string::npos)
	mode = true;
      else if(mode)
	{
	  if(line.find('%') != string::npos)
	    continue;
	  //First line containing row number, column number and NNS
	  else if(!ctr)
	    {
	      ss >> temp >> temp2 >> temp3;
	      NNS = temp3;
	      adj_list.resize(temp);
	      RS = temp;
	      ctr++;
	    }
      //Lines containing start and end of an edge
	  else if(ctr)
	    {
	      ss >> temp >> temp2;
	      if(temp != temp2)
		{
		  adj_list[temp - mini].push_back(temp2 - mini);
		  adj_list[temp2 - mini].push_back(temp - mini);
		}
	    }
	}
      //Get past comment lines
      else if(!mode)
	{
	  if(line.find('%') != string::npos)
	    continue;
	  //First line containing row number, column number and NNS
	  else if(!ctr)
	    {
	      ss >> temp >> temp2 >> temp3;
	      NNS = temp3;
	      adj_list.resize(temp);
	      rev_list.resize(temp);
	      RS = temp;
	      ctr++;
	    }
	  //Lines containing start and end of an edge
	  else if(ctr)
	    {
	      ss >> temp >> temp2;
	      if(temp != temp2)
		{
		  adj_list[temp - mini].push_back(temp2 - mini);
		  rev_list[temp2 - mini].push_back(temp - mini);
		}
	    }
	}
    }

  //Remove duplicates
  NNS = 0;
  NNSrev = 0;
  for(int i = 0; i < adj_list.size(); i++)
    {
      sort(adj_list[i].begin(), adj_list[i].end());
      adj_list[i].erase(unique(adj_list[i].begin(), adj_list[i].end()), adj_list[i].end());
      NNS += adj_list[i].size();
      if(!mode)
	{
	  sort(rev_list[i].begin(), rev_list[i].end());
	  rev_list[i].erase(unique(rev_list[i].begin(), rev_list[i].end()), rev_list[i].end());
	  NNSrev += rev_list[i].size();
	}
      
    }
  
  int* target_ind = (int*)(malloc(sizeof(int)*NNS));
  //cudaMallocHost(&target_ind, sizeof(int)*NNS);
  int* start_ind = (int*)(malloc(sizeof(int)*(RS+1)));
  //cudaMallocHost(&start_ind, sizeof(int)*(RS+1));
  int* results = (int*)(malloc(sizeof(int)*RS));
  //cudaMallocHost(&results, sizeof(int)*RS);
 
  memset(target_ind, 0, sizeof(int)* NNS);
  memset(start_ind, 0, sizeof(int)* (RS+1));
  memset(results, -1, sizeof(int)* RS);
  
  ConvertToCRS(target_ind, start_ind, adj_list, RS);

  int* target_rev;
  int* start_rev;
  if(!mode)
    {
      target_rev = (int*)(malloc(sizeof(int)*NNSrev));
      start_rev = (int*)(malloc(sizeof(int)*(RS+1)));
      memset(target_rev, 0, sizeof(int)* NNSrev);
      memset(start_rev, 0, sizeof(int)* (RS+1));
      ConvertToCRS(target_rev, start_rev, rev_list, RS);
    }
  
  
  //Start Node as parameter
  cout << "Graph converted to 0-base(Initial node is 0)\n";
  unsigned int startNode;
  //cout << "Please enter the start node: ";
  //cin >> startNode;
  //cout << endl;
  startNode = 0;

  if(mode)
    BFS(target_ind, start_ind, results, RS, NNS, startNode);
  else
    BFS_Directed(target_ind, start_ind, target_rev, start_rev, results, RS, NNS, NNSrev, startNode);

  ofstream myfile;
  myfile.open("hybridcudaresults.txt");
  for(int i = 0; i < RS; i++)
    myfile<< results[i] <<"\n";
  myfile.close();
  
  hipDeviceSynchronize();
  //cudaFreeHost(target_ind);
  //cudaFreeHost(start_ind);
  //cudaFreeHost(results);
  if(!mode)
    {
      free(target_rev);
      free(start_rev);
    }
  free(target_ind);
  free(results);
  free(start_ind);
  
  return 0;
}
