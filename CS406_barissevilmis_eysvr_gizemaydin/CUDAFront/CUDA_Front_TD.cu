
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <omp.h>
#include <stdlib.h>
#include <vector>
#include <stdio.h>
#include <string.h>
#include <queue>
#include <algorithm>
#include <cassert>

using namespace std;

void usage()
{
  cout << "USAGE: ./exec <filename> <deviceNum>" << endl;
  exit(0);
}

inline hipError_t checkCuda(hipError_t result) {
  if (result != hipSuccess) {
    fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    assert(result == hipSuccess);
  }
  return result;
}

void ConvertToCRS(int* & target_ind, int* & start_ind, const vector<vector<int> > &adj_list, int RS)
{
  int init = 0;
  for(int ii = 0; ii < RS ; ii++)
    { 
      start_ind[ii] = init;
      int siz = adj_list[ii].size();
      int where = 0;
      for(int jj = init; jj < init + siz; jj++)
	target_ind[jj] = adj_list[ii][where++];
      init += siz;
    }
  start_ind[RS] = init;
}

__global__
void BFS_Top_Down(int* target_ind, int* start_ind, int* results, int* frontier, int* frontsize, int* newfrontier, int* newfrontsize, int RS)
{
  int p = blockIdx.x * blockDim.x + threadIdx.x;
  
  if(p < (*frontsize))
    {
      int index = frontier[p];
      int start_loc = start_ind[index];
      int end_loc = start_ind[index + 1];
      int curr = results[index]; 
      for(int ii = start_loc; ii < end_loc; ii++)
	{
	  if(results[target_ind[ii]] == -1)
	    {
	      results[target_ind[ii]] = curr + 1;
	      
	      int cc = atomicAdd(newfrontsize,1);
	      newfrontier[cc] = target_ind[ii];
	      
	    }
	}
    }
}

__global__
void FRONT_COPY(int* front, int* newfront, int* newfrontsize)
{
  int ii = blockIdx.x * blockDim.x + threadIdx.x;
  if(ii < (*newfrontsize))
    {
      int index = newfront[ii];
      front[ii] = index;
    }
  
}

void BFS(int* & target_ind, int* & start_ind, int* & results, unsigned int RS, unsigned int NNS, unsigned int startNode)
{
  results[startNode] = 0;
  int *res;
  int *target_arr;
  int *start_arr;
  checkCuda(hipMalloc((void **)&res, sizeof(int)*RS));
  checkCuda(hipMalloc((void **)&target_arr, sizeof(int)*NNS));
  checkCuda(hipMalloc((void **)&start_arr, sizeof(int)*(RS+1)));
  //printf("Device Variable Copying:\t%s\n", cudaGetErrorString(cudaGetLastError()));

  int *size = (int*)(malloc(sizeof(int)));
  int *zero = (int*)(malloc(sizeof(int)));
  int *frontbeg = (int*)(malloc(sizeof(int)*RS));
  unsigned int CHUNK_SIZE = 1024;
  //unsigned int LIMIT = (RS+CHUNK_SIZE-1) / CHUNK_SIZE;
  dim3 GRID(1);
  dim3 BLOCK(CHUNK_SIZE);
  *size = 1;
  *zero = 0;
  memset(frontbeg, startNode, sizeof(int)*RS);
  
  //cudaMallocHost(&size,sizeof(int));
  int* frontsize;
  int* frontier;
  int* newfrontsize;
  int* newfrontier;
  int* tempptr;
  checkCuda(hipMalloc((void**)&frontier,sizeof(int)*RS));
  checkCuda(hipMalloc((void**)&frontsize,sizeof(int)));
  checkCuda(hipMalloc((void**)&newfrontier,sizeof(int)*RS));
  checkCuda(hipMalloc((void**)&newfrontsize,sizeof(int)));
  
  checkCuda(hipMemcpy(frontier,frontbeg,sizeof(int)*RS, hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(frontsize,size,sizeof(int), hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(newfrontsize,zero,sizeof(int), hipMemcpyHostToDevice));
  
  checkCuda(hipMemcpy(res, results, sizeof(int)*RS,hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(start_arr, start_ind, sizeof(int)*(RS+1),hipMemcpyHostToDevice));
  checkCuda(hipMemcpy(target_arr, target_ind, sizeof(int)*NNS,hipMemcpyHostToDevice));
  
  double start = omp_get_wtime();
  while(*size > 0)
    {

      BFS_Top_Down<<<GRID,BLOCK>>>(target_arr, start_arr, res, frontier, frontsize, newfrontier, newfrontsize, RS);
      hipDeviceSynchronize();
      hipMemcpy(size, newfrontsize, sizeof(int),hipMemcpyDeviceToHost);
      hipMemcpy(frontsize, size, sizeof(int),hipMemcpyHostToDevice);
      hipMemcpy(newfrontsize, zero, sizeof(int),hipMemcpyHostToDevice);
      tempptr = frontier;
      frontier = newfrontier;
      newfrontier = tempptr;
  
      GRID.x = (*size + 1023) / 1024;
    }
  
  double end = omp_get_wtime();
  cout << "\tSource:" << startNode << "\tTime:" << end - start << " s" << endl;
  checkCuda(hipMemcpy(results, res, sizeof(int)*RS,hipMemcpyDeviceToHost));

  free(size);
  free(zero);
  free(frontbeg);
  hipFree(frontier);
  hipFree(newfrontier);
  hipFree(frontsize);
  hipFree(newfrontsize);
  hipFree(res);
  hipFree(target_arr);
  hipFree(start_arr);
}

int main(int argc, const char** argv){

  if(argc != 3)
    usage();

  
  int devId = 0;
  devId = atoi(argv[2]); 
  hipSetDevice(devId); 
  string line;
  vector<vector<int> > adj_list;
  //See which node you are on by index+min
  
  const char* filename = argv[1];
  
  ifstream input (filename);
  if(input.fail())
    return 0;

  //Find first node
  unsigned int mini = 10000000;
  while(getline(input,line))
    {
      stringstream sf(line);
      int temp;
      if(line.find('%') != string::npos)
	continue;
      else
	{
	  while(sf >> temp)
	    {
	      if(temp < mini)
		mini = temp;
	    }
	}
    }
  //Set to begin
  input.clear();
  input.seekg(0, ios::beg);
  int ctr = 0;
  bool mode = false;
  unsigned int NNS, RS;
  while(getline(input,line))
    {
      stringstream ss(line);
      int temp, temp2, temp3;
      if(line.find("%%") != string::npos && line.find(" symmetric") != string::npos)
	mode = true;
      else if(mode)
	{
	  if(line.find('%') != string::npos)
	    continue;
	  //First line containing row number, column number and NNS
	  else if(!ctr)
	    {
	      ss >> temp >> temp2 >> temp3;
	      NNS = temp3;
	      adj_list.resize(temp);
	      RS = temp;
	      ctr++;
	    }
      //Lines containing start and end of an edge
	  else if(ctr)
	    {
	      ss >> temp >> temp2;
	      if(temp != temp2)
		{
		  adj_list[temp - mini].push_back(temp2 - mini);
		  adj_list[temp2 - mini].push_back(temp - mini);
		}
	    }
	}
      //Get past comment lines
      else if(!mode)
	{
	  if(line.find('%') != string::npos)
	    continue;
	  //First line containing row number, column number and NNS
	  else if(!ctr)
	    {
	      ss >> temp >> temp2 >> temp3;
	      NNS = temp3;
	      adj_list.resize(temp);
	      RS = temp;
	      ctr++;
	    }
	  //Lines containing start and end of an edge
	  else if(ctr)
	    {
	      ss >> temp >> temp2;
	      if(temp != temp2)
		adj_list[temp - mini].push_back(temp2 - mini);
	    }
	}
    }

  //Remove duplicates
  NNS = 0;
  for(int i = 0; i < adj_list.size(); i++)
    {
      sort(adj_list[i].begin(), adj_list[i].end());
      adj_list[i].erase(unique(adj_list[i].begin(), adj_list[i].end()), adj_list[i].end());
      NNS += adj_list[i].size();
    }
  
  int* target_ind = (int*)(malloc(sizeof(int)*NNS));
  //cudaMallocHost(&target_ind, sizeof(int)*NNS);
  int* start_ind = (int*)(malloc(sizeof(int)*(RS+1)));
  //cudaMallocHost(&start_ind, sizeof(int)*(RS+1));
  int* results = (int*)(malloc(sizeof(int)*RS));
  //cudaMallocHost(&results, sizeof(int)*RS);
 
  memset(target_ind, 0, sizeof(int)* NNS);
  memset(start_ind, 0, sizeof(int)* (RS+1));
  memset(results, -1, sizeof(int)* RS);
  
  ConvertToCRS(target_ind, start_ind, adj_list, RS);
  
  //Start Node as parameter
  cout << "Graph converted to 0-base(Initial node is 0)\n";
  unsigned int startNode;
  //cout << "Please enter the start node: ";
  //cin >> startNode;
  //cout << endl;
  startNode = 0;
  
  BFS(target_ind, start_ind, results, RS, NNS, startNode);

  ofstream myfile;
  myfile.open("cudafrontresults.txt");
  for(int i = 0; i < RS; i++)
    myfile<< results[i] <<"\n";
  myfile.close();
  
  hipDeviceSynchronize();
  //cudaFreeHost(target_ind);
  //cudaFreeHost(start_ind);
  //cudaFreeHost(results);
  free(target_ind);
  free(results);
  free(start_ind);
  
  return 0;
}
